#include "hip/hip_runtime.h"

#include <assert.h>
#include <iostream>
# include <time.h>
# include <stdlib.h>
# include <stdio.h>
# include <string.h>
# include <math.h>
# include <hip/hip_runtime.h>
# include <ctime>
#include <fstream>
#include <sstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include </content/read.h>

#define BLOCK_SIZE 16

/*
*********************************************************************
function name: gpu_matrix_mult
description: dot product of two matrix (not only square)
parameters: 
            &a GPU device pointer to a m X n matrix (A)
            &b GPU device pointer to a n X k matrix (B)
            &c GPU device output purpose pointer to a m X k matrix (C) 
            to store the result
Note:
    grid and block should be configured as:
        dim3 dimGrid((k + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    further sppedup can be obtained by using shared memory to decrease global memory access times
return: none
*********************************************************************
*/
__constant__ float width =18; // user input - width of the enclosure
__constant__ float height = 19.5; // user input - height of the enclosure
__constant__ int image_width = 64; // user input
__constant__ int image_height = 64;
__constant__ int transimitter_pos;

__global__ void add(int *d_a, int *d_c, int *d_result, int n) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;
    d_result[row * n + col] = d_result[row * n + col] + d_c[row * n + col];       
}
__global__ void gpu_matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

/*
*********************************************************************
function name: gpu_square_matrix_mult
description: dot product of two matrix (not only square) in GPU
parameters: 
            &a GPU device pointer to a n X n matrix (A)
            &b GPU device pointer to a n X n matrix (B)
            &c GPU device output purpose pointer to a n X n matrix (C) 
            to store the result
Note:
    grid and block should be configured as:
        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
return: none
*********************************************************************
*/
__global__ void total_focusing(int *d_a, int *d_b, int *d_result, int n,int **arrays, int pixels,int x, int y) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;
    int pixel_width =width/pixels; // in constant memory
    int x_cordinate = col*pixel_width; // x coordinate of the pixel on the grid
    int y_cordinate = row*pixel_width; // y coordinate of the pixel on the grid
    //float transmitter_distance = sqrt(pow(x -  x_cordinate, 2) +pow(y -  y_cordinate, 2) + pow(height - 0, 2) * 1.0);// distance of transmitter to pixel grid position
    float transmitter_distance = sqrt(pow(x -  x_cordinate, 2) +pow(y -  y_cordinate, 2));
    // x and y are coordinates of the receiver
     
    float intensity = 0; // intensity of the pixel where each child is going to add  amplitude of the Ascan
    
    for (int j = 0; j < 8; j++){

         int  y1 = 7.5; // for linear configuration
         int  x1 = 0;
          if ( j == 0){
            //x1 = 3.1;
              x1 = 11;
          }
          else if (j == 1){
              //x1 = 7.3;
              x1 = 12;
          }
          else if (j == 2){
              //x1 = 11.5;
              x1   =12.5;
          }
          else{
              x1 = 15.7;
            }

         //TODO: find the cordinates of the reciever  ???

         //TODO: calculate the distance from the receiver to pixel
         int receiver_pos = j+1;
         //float receiver_distance = sqrt(pow(x1 -  x_cordinate, 2) +pow(y1 -  y_cordinate, 2) + pow(height - 0, 2) * 1.0);// distance of receiver to pixel grid position
         float receiver_distance = sqrt(pow(x1 -  x_cordinate, 2) +pow(y1 -  y_cordinate, 2));
         // TODO: calculate time of fligtht to the pixel position in microseconds
         // Distance formula is 0.034cm/microsecond x time of flight

         int time_of_flight = (receiver_distance + transmitter_distance)/0.034;

         //TODO: extract the amplitude for the A-scan  at the calculated time of flight
          float amplitude = 0;

          // outuput[row*width + col] = sum.
         // N is width, i is row
         amplitude = arrays[j*4 + transimitter_pos][time_of_flight];
         intensity = intensity + amplitude;
         }

      //d_result[row * n + col] = arrays[2][2];
      d_result[row * n + col] = intensity;   
}

/*
*********************************************************************
function name: gpu_matrix_transpose
description: matrix transpose
parameters: 
            &mat_in GPU device pointer to a rows X cols matrix
            &mat_out GPU device output purpose pointer to a cols X rows matrix 
            to store the result
Note:
    grid and block should be configured as:
        dim3 dim_grid((n - 1) / BLOCK_SIZE + 1, (n - 1) / BLOCK_SIZE + 1, 1);
        dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);
return: none
*********************************************************************

*********************************************************************
function name: cpu_matrix_mult
description: dot product of two matrix (not only square) in CPU, 
             for validating GPU results
parameters: 
            &a CPU host pointer to a m X n matrix (A)
            &b CPU host pointer to a n X k matrix (B)
            &c CPU host output purpose pointer to a m X k matrix (C) 
            to store the result
return: none
*********************************************************************
*/
void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

/*
*********************************************************************
function name: main
description: test and compare
parameters: 
            none
return: none
*********************************************************************
*/
int main(int argc, char const *argv[])
{
   float p_width = 18; // user input - width of the enclosure
   float p_height = 19.5; // user input - height of the enclosure
   int p_image_width =64; // user input
   int p_image_height =64;
   int pixels =12; // number of image pixel. It must be a multiple of 32
   int p_N = 4; 

    int m, n, k;
    /* Fixed seed for illustration */
    srand(3333);
    printf("please type in m n and k\n");
    scanf("%d %d %d", &m, &n, &k);

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);

    // random initialize matrix A -low reso
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = 0;
        }
    }

    // random initialize matrix B -high reso
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = 0;
        }
    }


    const int N_ARRAYS = 16;
    int *arrayOfArrays[N_ARRAYS];
    int *darrayOfArrays[N_ARRAYS];
    int **d_array;
   // int arr_len[N_ARRAYS] = {3, 2, 3};
    int arr1[3] = {1,2,3};
    //for (int k = 0; k < 16;k++){
      // arrayOfAscans[k] = arr1;
    // }
    
    int array1[70000];
    int array2[70000];
    int array3[70000];
    int array4[70000];
    int array5[70000];
    int array6[70000];
    int array7[70000];
    int array8[70000];
    int array9[70000];
    int array10[70000];
    int array11[70000];
    int array12[70000];
    int array13[70000];
    int array14[70000];
    int array15[700000];
    int array16[80000];

    read(array1,array2,array3,array4,array5,array6,array7,array8,array9,array10,array11,array12,array13,array14,array15,array16);
    arrayOfArrays[0] = array1;
    arrayOfArrays[1] = array2;
    arrayOfArrays[2] = array3;
    arrayOfArrays[3] = array4;
    arrayOfArrays[4] = array5;
    arrayOfArrays[5] = array6;
    arrayOfArrays[6] = array7;
    arrayOfArrays[7] = array8;
    arrayOfArrays[8] = array9;
    arrayOfArrays[9] = array10;
    arrayOfArrays[10] = array11;
    arrayOfArrays[11] = array12;
    arrayOfArrays[12] = array13;
    arrayOfArrays[13] = array14;
    arrayOfArrays[14] = array15;
    arrayOfArrays[15] = array16;
   
    // 1) You have to allocate the pointers to a host memory,
    for(int i = 0; i < N_ARRAYS; i++){
        //2) then allocate device memory for each array
        hipMalloc(&(darrayOfArrays[i]), 70000* sizeof(int));
        hipMemcpy(darrayOfArrays[i], arrayOfArrays[i], 70000*sizeof(int), hipMemcpyHostToDevice); // copy contents of each array to device
    }

    // 4) Allocate the memmory for storing the pointers into the device to *d_array
    hipMalloc(&d_array, sizeof(int*) * N_ARRAYS);

    // 5) Copy arrayOfArrays to d_array of size sizeof(void*) * N_ARRAYS from Host to device
    hipMemcpy(d_array, darrayOfArrays, sizeof(int*) * N_ARRAYS, hipMemcpyHostToDevice);

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
    // Allocate memory space on the device 
    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
    for (int i = 0; i < 4; i++) {

    int x1 = 0;
    int y1 = 7.5;

      if ( i == 0){
            x1 = 1;
          }
       else if (i == 1){
              x1 = 5.2;
          }
       else if (i == 2){
              x1 = 9.4;
          }
        else{
              x1 = 13.6;
            }
    int transimitter_pos_1 = i; // this variable is used to create the file name to access
    hipMemcpyToSymbol(HIP_SYMBOL(transimitter_pos), &transimitter_pos_1, sizeof(int));

    // Launch kernel 
    if(m == n && n == k)
    {
        total_focusing<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n,d_array,pixels,x1,y1);    
    }
    else
    {
        gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_c, d_b, m, n, k);    
    }

    add<<<dimGrid,dimBlock>>>(d_a, d_c, d_b,n);

     }

    // Transefr results from device to host 
    //hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipMemcpy(h_c, d_b, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);
    
     printf("\n");
    // start the CPU version
    hipEventRecord(start, 0);

    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n\n", m, n, n, k, cpu_elapsed_time_ms);

    // validate results computed by GPU
    int all_ok = 1;

    ofstream image;
    image.open("smoll2.ppm");
    if (image.is_open()){
    // TODO: Place header infor
    image<<"P3" << endl;
    image<<"64 64" <<endl; // size of pixels
    image<<"255"<<endl;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            //printf("[%d][%d]:%d == [%d][%d]:%d, ", i, j, h_cc[i*k + j], i, j, h_c[i*k + j]);
            //printf("%d\t",h_c[i*k + j]);
            //image<<(h_c[i*k + j]*10)%250 <<"  "<<(h_c[i*k + j]*1)%250 << " "<<(h_c[i*k + j]*1)%250<< endl;
            if (h_c[i*k + j]%250 <= 50 )
            {
              image<< 29 <<"  "<< 246<< " "<< 246 << endl;
            }
            
            if ((h_c[i*k + j]%250 <=100) && (h_c[i*k + j]%250 > 50))
            {
              image<< 29 <<"  "<< 246<< " "<< 116 << endl;
            }
            if ((h_c[i*k + j]%250 <=120) && (h_c[i*k + j]%250 > 100))
            {
              image<< 12 <<"  "<< 209<< " "<< 91 << endl;
            }
            if ((h_c[i*k + j]%250 <= 150) && (h_c[i*k + j]%250 > 120))
            {
              image<< 12 <<"  "<< 240<< " "<< 27 << endl;
            }
            if ((h_c[i*k + j]%250 <=180) && (h_c[i*k + j]%250 > 150))
            {
              image<< 218 <<"  "<< 240<< " "<< 12 << endl;
            }
            if ((h_c[i*k + j]%250 <=200) && (h_c[i*k + j]%250 > 180))
            {
              image<< 245 <<"  "<< 237<< " "<< 12 << endl;
            }
            if ((h_c[i*k + j]%250 <=220) && (h_c[i*k + j]%250 > 200))
            {
              image<< 245 <<"  "<< 105<< " "<< 12 << endl;
            }
            if ((h_c[i*k + j]%250 <= 250) && (h_c[i*k + j]%250 > 220))
            {
              image<< 245 <<"  "<< 28<< " "<< 12 << endl;
            }
              
            if(h_cc[i*k + j] != h_c[i*k + j])
            {
                all_ok = 0;
            }
        }
        printf("\n");
    }
   image.close();

    // roughly compute speedup
    if(all_ok)
    {
        printf("all results are correct!!!, speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    }
    else
    {
        printf("incorrect results\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}}
