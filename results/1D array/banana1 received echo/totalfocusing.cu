#include "hip/hip_runtime.h"
// Dean Makoni
// total focusing method
#include <iostream>
# include <time.h>
# include <stdlib.h>
# include <stdio.h>
# include <string.h>
# include <math.h>
# include <hip/hip_runtime.h>
# include <ctime> 
#include <fstream>
#include <sstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include </home/MKNDEA002/final_year/src/read.h>
#include <iostream>

using namespace std;

  __constant__ float width =18; // user input - width of the enclosure
  __constant__ float height = 16; // user input - height of the enclosure
  __constant__ int image_width = 64; // user input
  __constant__ int image_height = 64; //  user input
  __constant__ int N = 4;  // number of receivers
  __constant__ int transimitter_pos;
  __constant__ float x; // x cordinate of  transmitter
  __constant__ float y; // y cordinate of the  transmitter
  __constant__ float z;



__global__ void add( unsigned char * arr,unsigned char * arr2) {
   
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
//  arr2[idy* image_width + idx ] = arr[idy* image_width + idx ] + arr2[idy* image_width + idx ]; 
    arr2[idy* image_width + idx ] = 0;
} 

__global__ void pixelKernel(unsigned char * low_reso, int pixels,float x,float y, float z,int **Ascans)
{
  // width and height are the dimensions of the enclosure. They are not supposed to be confused with the dimensions of the image
  float threadID = threadIdx.x;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;  ///  ??????????????
  int pixel_width =width/pixels; // in constant memory
  int x_cordinate = idx*pixel_width; // x coordinate of the pixel on the grid
  int y_cordinate = idy*pixel_width; // y coordinate of the pixel on the grid
  float transmitter_distance = sqrt(pow(x -  x_cordinate, 2) +pow(y -  y_cordinate, 2) + pow(height - 0, 2) * 1.0);// distance of transmitter to pixel grid position
  // x and y are coordinates of the receiver
 
  float intensity = 0; // intensity of the pixel where each child is going to add  amplitude of the Ascan 

  //childKernel<<<1,receivers>>>(x,y,transmitter_pos);// number of child kernels are determined by number of receicers
  // use one block here for threads
  // weight the option of using cuda streams
   using namespace std;
   using std::string;
  
  for (int j = 0; j < N; j++){

         int  y1 = 2.5; // for linear configuration
         int  x1 = 0;
          if ( j == 0){
            x1 = 3.1;
          }
          else if (j == 1){
              x1 = 7.3;
          }
          else if (j == 2){
              x1 = 11.5;
          }
          else{
              x1 = 15.7;
            }

  	 //TODO: find the cordinates of the reciever  ???

  	 //TODO: calculate the distance from the receiver to pixel
         int receiver_pos = j+1;
  	 float receiver_distance = sqrt(pow(x1 -  x_cordinate, 2) +pow(y1 -  y_cordinate, 2) + pow(height - 0, 2) * 1.0);// distance of receiver to pixel grid position

  	 // TODO: calculate time of fligtht to the pixel position in microseconds
  	 // Distance formula is 0.034cm/microsecond x time of flight

  	 int time_of_flight = (receiver_distance + transmitter_distance)/0.034;

  	 //TODO: extract the amplitude for the A-scan  at the calculated time of flight
          float amplitude = 0;
        
          // outuput[row*width + col] = sum.
         // N is width, i is row
          Ascans[j*N + transimitter_pos][time_of_flight] = amplitude;  
          intensity = intensity + amplitude;
   	 }
    __syncthreads();
   
  // place the intensity to proper position in low resolution image
   low_reso[idy* image_width + idx ] = intensity;
  // idx = col
  // idy = row
  // image_width = width
}

int main(void)
{ 
   
   float p_width = 16.5; // user input - width of the enclosure
   float p_height = 16; // user input - height of the enclosure
   int p_image_width =64; // user input
   int p_image_height =64;
   int pixels =64; // number of image pixel. It must be a multiple of 32
   int p_N = 4; // number of transimitters user input 
  
 // printf("dean");
  hipMemcpyToSymbol(HIP_SYMBOL(width), &p_width, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(height), &p_height, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(image_width), &p_image_width, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(image_height), &p_image_height, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(N),&p_N, sizeof(int));

  unsigned char * high_reso; // high resolution image 
  unsigned char * high_reso_device; // high resolution device in device memory  copy it to high reso imeage after all the updates
  
  int ImageSize = sizeof ( unsigned char ) *64 * 64 ;
  hipMalloc (( void **) & high_reso_device, ImageSize );
  high_reso = ( unsigned char *) malloc ( ImageSize ); // allocate image a host memory
  /** 
    Populate the Ascan  array. By copying Ascan array in data directory.
    Basically making an array of arrays. 
   **/
    const int N_ARRAYS = 16;
    int *arrayOfAscans[N_ARRAYS];  // arrayOfArrays
    int *darrayOfAscans[N_ARRAYS]; 
    int **d_array;
   // int arr_len[N_ARRAYS] = {3, 2, 3};
  //  int arr1[3] = {1,2,3};

    int array1[7000];
    int array2[7000];
    int array3[7000];
    int array4[7000];
    int array5[7000];
    int array6[7000];
    int array7[7000];
    int array8[7000];
    int array9[7000];
    int array10[7000];
    int array11[7000];
    int array12[7000];
    int array13[7000];
    int array14[7000];
    int array15[7000];
    int array16[7000];

    read(array1,array2,array3,array4,array5,array6,array7,array8,array9,array10,array11,array12,array13,array14,array15,array16);
    arrayOfAscans[0] = array1;
    arrayOfAscans[1] = array2;
    arrayOfAscans[2] = array3;
    arrayOfAscans[3] = array4;
    arrayOfAscans[4] = array5;
    arrayOfAscans[5] = array6;
    arrayOfAscans[6] = array7;
    arrayOfAscans[7] = array8;
    arrayOfAscans[8] = array9;
    arrayOfAscans[9] = array10;
    arrayOfAscans[10] = array11;
    arrayOfAscans[11] = array12;
    arrayOfAscans[12] = array13;
    arrayOfAscans[13] = array14;
    arrayOfAscans[14] = array15;
    arrayOfAscans[15] = array16;
         
   // printf("dean2");
   // for (int k = 0; k < 20;k++){
   //	 printf("%d", array1[k]);
     //    printf( "\n" );
   //  }
   
   // allocating pointers to host memory 
   for(int i = 0; i < N_ARRAYS; i++){
        //Allocating device memory for each array
        hipMalloc(&(darrayOfAscans[i]), 1000000 * sizeof(float));
        hipMemcpy(darrayOfAscans[i], arrayOfAscans[i], 1000000*sizeof(float), hipMemcpyHostToDevice); // copy contents of each array to device
    }   
  //Allocating the memmory for storing the pointers into the device to *d_array
  hipMalloc(&d_array, sizeof(float*) * N_ARRAYS);
  
  //Copy arrayOfAscans to d_array of size sizeof(void*) * N_ARRAYS from Host to device
    hipMemcpy(d_array, darrayOfAscans, sizeof(float*) * N_ARRAYS, hipMemcpyHostToDevice);

/** iterate through all the receivers to get low resolution images
 First step is to calculate  the transmitter position
 receiver coordinates are supposd to be kept in constant memory - constant memory is designed for faster parallel data access 
 first[B step copy high resolution image to device memory
 after every iteration the iterations adds its intensities to high resolution image
 copy the image to host after all the iterations.
 **/

  int x1 = 0;
  int y1 = 2.5;
  for (int i = 0; i < p_N; i++) {

      if ( i == 0){
            x1 = 1;
          }
       else if (i == 1){
              x1 = 5.2;
          }
       else if (i == 2){
              x1 = 9.4;
          }
        else{
              x1 = 13.6;
            }
   
    int transimitter_pos_1 = i; // this variable is used to create the file name to access
    hipMemcpyToSymbol(HIP_SYMBOL(transimitter_pos), &transimitter_pos_1, sizeof(int)); 

    unsigned char * low_reso_device = NULL; // low resolution image on device
    hipMalloc (( void **) & low_reso_device , ImageSize );
    
    //TODO: launch parent kernel

    pixelKernel<<<pixels/32,pixels>>>(low_reso_device,pixels,x1,y1,16,d_array); // pixels dived by 32 gives the number of blocks
    hipDeviceSynchronize();
    //printf("dean4");
   //TODO: launch a kernel that adds the results of high_reso _device and low_reso_device
    add<<<pixels/32, pixels>>>(low_reso_device,high_reso_device);
    hipDeviceSynchronize();
    hipFree(low_reso_device);
  }

 // TODO ;Copy the high_reso_device to high_reso on the CPU
  hipMemcpy ( high_reso, high_reso_device, ImageSize ,hipMemcpyDeviceToHost );

 // TODO: Print the high resolution  image on CPU
    

    // ofstream image;
    // image.open("test3.ppm");
     // if (image.is_open()){
     // TODO: Place header infor
    // image<<"P3" << endl;
    // image<<"64 64" <<endl; // size of pixels
    // image<<"255"<<endl;
    for(int i=0;i<64;i++)
    {
      printf("\n");
      for(int j=0;j <64; j++){
          printf("%d\t", high_reso[i* 64 + j]);
           //image<<(high_reso[i* 64 + j]*10)%4096 <<"  "<<(high_reso[i* 64 + j]*3)%4096 << " "<<(high_reso[i* 64 + j]*5)%4096<< endl;
          // image<< i*2 << " " << i << " "<< i << endl;
     }
    // printf("\n");
     }
 // }
  // image.close();
  //TODO:  Free memory

  hipFree(high_reso_device);

    for(int i = 0; i < N_ARRAYS; i++){
        hipFree(darrayOfAscans[i]); //host not device memory
        //TODO: check error
    }
    hipFree(d_array);
    // free(arrayOfArrays);
    //printf("%s\n", hipGetErrorString(hipGetLastError()));  
    //printf("dean5");
  return 0;
}
